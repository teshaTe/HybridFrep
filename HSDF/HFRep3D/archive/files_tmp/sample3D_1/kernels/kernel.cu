#include "hip/hip_runtime.h"
﻿#include "kernel.h"
#include "device_funcs.cuh"
#include "helper_cuda_math.h"

#include <stdio_ext.h>

#define TX_2D 32
#define TY_2D 32
#define TX 8
#define TY 8
#define TZ 8

#define NUMSTEPS 20

__global__ void renderKernel( uchar4 *d_out, float *d_vol, int w, int h, int3 volSize,
                              int method, float zs, float theta, float threshold, float dist )
{
    const int col = blockIdx.x*blockDim.x + threadIdx.x;
    const int row = blockIdx.y*blockDim.y + threadIdx.y;
    const int i   = col + row * w;

    if( ( col >= w ) || ( row >= h ) ) return; //checking if still within image bounds

    const uchar4 background = { 64, 0, 128, 0 };
    float3 source = { 0.0f, 0.0f, -zs };
    float3 pix    = srcIdxToPos( col, row, w, h, 2*volSize.z - zs );

    //apply viewing transformation: rotate around y-axis
    source = yRotate( source, theta );
    pix    = yRotate( pix, theta );

    //prepare inputs for ray-box intersection
    float t0, t1;
    const Ray pixRay = { source, pix - source };
    float3 center    = { volSize.x/2.0f, volSize.y/2.0f, volSize.z/2.0f };

    const float3 boxmin = -center;
    const float3 boxmax = { volSize.x - center.x, volSize.y - center.y, volSize.z - center.z };

    //perform ray-box intersection test
    const bool hitBox = intersectBox( pixRay, boxmin, boxmax, &t0, &t1 );
    uchar4 shade;
    if( !hitBox )
        shade = background;          // miss box -> background color
    else
    {
        if( t0 < 0.0f )  t0 = 0.0f;  // clamp to 0 to avoid looking backward

        //bounded by the points where the ray enters and leaves the box
        const Ray boxRay = { paramRay( pixRay, t0 ), paramRay( pixRay, t1 ) - paramRay( pixRay, t0 ) };

        if( method == 1 )
            shade = sliceShader( d_vol, volSize, boxRay, threshold, dist, source );
        else if ( method == 2 )
            shade = rayCastShader( d_vol, volSize, boxRay, threshold );
        else
            shade = volumeRenderShader( d_vol, volSize, boxRay, threshold, NUMSTEPS );
    }

    d_out[i] = shade;
}

/*__global__ void volumeKernel( float *d_vol, int3 volSize, int id, float4 params )
{
    const int w     = volSize.x, h = volSize.y, d = volSize.z;

    const int col   = blockIdx.x * blockDim.x + threadIdx.x;
    const int row   = blockIdx.y * blockDim.y + threadIdx.y;
    const int stack = blockIdx.z * blockDim.z + threadIdx.z;
    const int i     = col + row * w + stack * w * h;

    if ( ( col >= w ) || ( row >= h ) || ( stack >= d ) ) return;

    d_vol[i] = func( col, row, stack, id, volSize, params ); //computing and store the volume
}*/

__global__ void distanceTrkernel( float *d_frep, float *d_udf, int3 fieldSize )
{

}

__global__ void volumeKernel( float *d_sdf, float *d_vol, int3 volSize, int3 sdfSize )
{
    const int w = volSize.x, h = volSize.y, d = volSize.z;

    const int col   = blockIdx.x * blockDim.x + threadIdx.x;
    const int row   = blockIdx.y * blockDim.y + threadIdx.y;
    const int stack = blockIdx.z * blockDim.z + threadIdx.z;
    const int i     = col + row * w + stack * w * h;

    if ( ( col >= w ) || ( row >= h ) || ( stack >= d ) ) return;

    float3 k = make_float3( volSize.x / sdfSize.x, volSize.y / sdfSize.y, volSize.z / sdfSize.z );

    float3 pos = { float(col/k.x), float(row/k.y), float(stack/k.z)};
    d_vol[i] = interpolate_in_volume( d_sdf, sdfSize, pos ); //computing and store the volume
}

void kernelLauncher( uchar4 *d_out, float *d_vol, int w, int h, int3 volSize,
                     int method, int zs, float theta, float threshold, float dist )
{
    dim3 blockSize( TX_2D, TY_2D );
    dim3 gridSize( divUp( w, TX_2D ), divUp( h, TY_2D ) );
    renderKernel <<< gridSize, blockSize >>> ( d_out, d_vol, w, h, volSize, method, zs, theta, threshold, dist );
}

/*void volumeKernelLauncher( float *d_vol, int3 volSize, int id, float4 params )
{
    dim3 blockSize( TX, TY, TZ );
    dim3 gridSize( divUp( volSize.x, TX ), divUp( volSize.y, TY ), divUp( volSize.z, TZ ) );
    volumeKernel <<< gridSize, blockSize >>> ( d_vol, volSize, id, params );
}*/

void volumeKernelLauncher(float *h_sdf, float *d_sdf, float *d_vol, int3 volSize , int3 sdfSize)
{
    dim3 blockSize( TX, TY, TZ );
    dim3 gridSize( divUp( volSize.x, TX ), divUp( volSize.y, TY ), divUp( volSize.z, TZ ) );

    hipMemcpy( d_sdf, h_sdf, float(sdfSize.x*sdfSize.y*sdfSize.z)*sizeof(float), hipMemcpyHostToDevice );

    volumeKernel <<< gridSize, blockSize >>> ( d_sdf, d_vol, volSize, sdfSize );
}


void distanceTrKernelLauncher( float *h_frep, float *d_frep, float *d_udf, int3 fieldSize )
{
    dim3 blockSize ( TX, TY, TZ );
    dim3 gridSize( divUp( fieldSize.x, TX ), divUp( fieldSize.y, TY ), divUp( fieldSize.z, TZ ) );

    hipMemcpy( d_frep, h_frep, float( fieldSize.x*fieldSize.y*fieldSize.z )*sizeof(float), hipMemcpyHostToDevice );
    distanceTrkernel <<< gridSize, blockSize >>> ( d_frep, d_udf, fieldSize );
}

