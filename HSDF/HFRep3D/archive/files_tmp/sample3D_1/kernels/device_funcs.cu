#include "hip/hip_runtime.h"
#include "device_funcs.cuh"
#include "helper_cuda_math.h"

#include <stdio.h>

#define EPS 0.01f

__host__ int divUp( int a, int b ) { return ( a + b - 1) / b; }

__device__ unsigned char clip( int n ) { return n > 255 ? 255 : ( n < 0 ? 0 : n ); }

__device__ int clipWithBounds( int n, int n_min, int n_max )
{
    return n > n_max ? n_max : ( n < n_min ? n_min : n );
}


__device__ float3 xRotate( float3 pos, float theta )
{
    const float c = cosf( theta ), s = sinf( theta );
    return make_float3( pos.x, c*pos.y - s*pos.z, s*pos.y + c*pos.z );
}

__device__ float3 yRotate( float3 pos, float theta )
{
    const float c = cosf( theta ), s = sinf( theta );
    return make_float3( c*pos.x + s*pos.z, pos.y, -s*pos.x + c*pos.z );
}

__device__ float3 zRotate( float3 pos, float theta )
{
    const float c = cosf( theta ), s = sinf( theta );
    return make_float3( c*pos.x - s*pos.y, s*pos.x + c*pos.y, pos.z );
}


/*__device__ float func( int col, int row, int stack, int id, int3 volSize, float4 params )
{
    const int3 pos0 = { volSize.x/2, volSize.y/2, volSize.z/2 };
    //const float dx = float(col - pos0.x)/float(volSize.x),
    //            dy = float(row - pos0.y)/float(volSize.y),
    //            dz = float(stack - pos0.z)/float(volSize.z);

    const float dx = float(col - pos0.x),
                dy = float(row - pos0.y),
                dz = float(stack - pos0.z);

    //sphere
    if( id == 0 )
    {
        return sqrtf( dx*dx + dy*dy + dz*dz ) - params.x ; //r = 0.36 if not params
    }
    //torus
    else if ( id == 1 )
    {
        const float r = sqrtf( dx*dx + dy*dy );
        return sqrtf( ( r - params.x )*( r - params.x ) + dz*dz ) - params.y;
    }
    //block
    else
    {
        float x = fabsf( dx ) - params.x, y = fabsf( dy ) - params.y, z = fabsf( dz ) - params.z;
        if( x <= 0 && y <= 0 && z <= 0 )
            return fmaxf( x, fmaxf( y, z ) );
        else
        {
            x = fmaxf( x, 0 ), y = fmaxf( y, 0 ), z = fmaxf( z, 0 );
            return sqrtf( x*x + y*y + z*z );
        }
    }
}*/

__device__ float3 srcIdxToPos( int col, int row, int w, int h, float zs )
{
    return make_float3( col - w/2, row - h/2, zs );
}

__device__ float3 paramRay( Ray r, float t ) { return r.o + t *(r.d); }

__device__ float planeSDF( float3 pos, float3 norm, float d )
{
    return dot( pos, normalize( norm ) ) - d;
}

__device__ bool rayPlaneIntersect( Ray myRay, float3 n, float dist, float *t )
{
    const float f0 = planeSDF( paramRay( myRay, 0.0f ), n, dist );
    const float f1 = planeSDF( paramRay( myRay, 1.0f ), n, dist );
    bool result = ( f0 * f1 < 0 );
    if( result ) *t = ( 0.0f - f0 ) / ( f1 - f0 );
    return result;
}

__device__ bool intersectBox( Ray r, float3 boxmin, float3 boxmax, float *tnear, float *tfar )
{
    //compute intersection of ray with all six bbox planes
    const float3 invR = make_float3( 1.0f ) / r.d;
    const float3 tbot = invR * ( boxmin - r.o ), ttop = invR * (boxmax - r.o );

    //Re-order intersections to find smallest and largest on each axiss
    const float3 tmin = fminf( ttop, tbot ), tmax = fmaxf( ttop, tbot );

    //find the largest tmin and the smallest tmax
    *tnear = fmaxf( fmaxf( tmin.x, tmin.y ), fmaxf( tmin.x, tmin.z ) );
    *tfar  = fminf( fminf( tmax.x, tmax.y ), fminf( tmax.x, tmax.z ) );
    return *tfar > *tnear;
}

__device__ int3 posToVolIndex( float3 pos, int3 volSize )
{
    return make_int3( pos.x + volSize.x/2, pos.y + volSize.y/2, pos.z + volSize.z/2 );
}

__device__ int flatten( int3 index, int3 volSize )
{
    return index.x + index.y * volSize.x + index.z * volSize.x * volSize.y;
}

__device__ float interpolate_in_volume( float *d_sdf, int3 sdfSize, float3 pos )
{
    int3 index = { pos.x, pos.y, pos.z };
    int i = index.x, j = index.y, k = index.z;

    const int w  = sdfSize.x, h = sdfSize.y, d = sdfSize.z;
    const float3 rem = fracf(pos);

    index = make_int3( clipWithBounds( i, 0, w - 2 ),
                       clipWithBounds( j, 0, h - 2 ),
                       clipWithBounds( k, 0, d - 2 ) );

    //directed incremenets for computing the gradient
    const int3 dx = { 1, 0, 0 }, dy = { 0, 1, 0 }, dz = { 0, 0, 1 };

    //directed increments for computing grid points
    const float dens000 = d_sdf[ flatten( index, sdfSize ) ];
    const float dens100 = d_sdf[ flatten( index + dx, sdfSize ) ];
    const float dens010 = d_sdf[ flatten( index + dy, sdfSize ) ];
    const float dens001 = d_sdf[ flatten( index + dz, sdfSize ) ];
    const float dens110 = d_sdf[ flatten( index + dx + dy, sdfSize ) ];
    const float dens101 = d_sdf[ flatten( index + dx + dz, sdfSize ) ];
    const float dens011 = d_sdf[ flatten( index + dy + dz, sdfSize ) ];
    const float dens111 = d_sdf[ flatten( index + dx + dy + dz, sdfSize ) ];

    //trilinear interpolation
    return ( 1 - rem.x ) * ( 1 - rem.y ) * ( 1 - rem.z ) * dens000 +
           rem.x         * ( 1 - rem.y ) * ( 1 - rem.z ) * dens100 +
           ( 1 - rem.x ) * rem.y         * ( 1 - rem.z ) * dens010 +
           ( 1 - rem.x ) * ( 1 - rem.y ) * rem.z         * dens001 +
           rem.x         * rem.y         * ( 1 - rem.z ) * dens110 +
           rem.x         * ( 1 - rem.y ) * rem.z         * dens101 +
           ( 1 - rem.x ) * rem.y         * rem.z         * dens011 +
            rem.x        * rem.y         * rem.z         * dens111;
}


__device__ float density( float *d_vol, int3 volSize, float3 pos )
{
    int3 index = posToVolIndex( pos, volSize );
    int i = index.x, j = index.y, k = index.z;

    const int w  = volSize.x, h = volSize.y, d = volSize.z;
    const float3 rem = fracf( pos );

    index = make_int3( clipWithBounds( i, 0, w - 2 ),
                       clipWithBounds( j, 0, h - 2 ),
                       clipWithBounds( k, 0, d - 2 ) );

    //directed incremenets for computing the gradient
    const int3 dx = { 1, 0, 0 }, dy = { 0, 1, 0 }, dz = { 0, 0, 1 };

    //directed increments for computing grid points
    const float dens000 = d_vol[ flatten( index, volSize ) ];
    const float dens100 = d_vol[ flatten( index + dx, volSize ) ];
    const float dens010 = d_vol[ flatten( index + dy, volSize ) ];
    const float dens001 = d_vol[ flatten( index + dz, volSize ) ];
    const float dens110 = d_vol[ flatten( index + dx + dy, volSize ) ];
    const float dens101 = d_vol[ flatten( index + dx + dz, volSize ) ];
    const float dens011 = d_vol[ flatten( index + dy + dz, volSize ) ];
    const float dens111 = d_vol[ flatten( index + dx + dy + dz, volSize ) ];

    //trilinear interpolation
    return ( 1 - rem.x ) * ( 1 - rem.y ) * ( 1 - rem.z ) * dens000 +
           rem.x         * ( 1 - rem.y ) * ( 1 - rem.z ) * dens100 +
           ( 1 - rem.x ) * rem.y         * ( 1 - rem.z ) * dens010 +
           ( 1 - rem.x ) * ( 1 - rem.y ) * rem.z         * dens001 +
           rem.x         * rem.y         * ( 1 - rem.z ) * dens110 +
           rem.x         * ( 1 - rem.y ) * rem.z         * dens101 +
           ( 1 - rem.x ) * rem.y         * rem.z         * dens011 +
            rem.x        * rem.y         * rem.z         * dens111;
}

__device__ uchar4 sliceShader( float *d_vol, int3 volSize, Ray boxRay, float gain, float dist, float3 norm )
{
    float t;
    uchar4 shade = make_uchar4( 96, 0, 192, 0 );
    if( rayPlaneIntersect( boxRay, norm, dist, &t ) )
    {
        float sliceDens = density( d_vol, volSize, paramRay( boxRay, t ) );
        shade = make_uchar4( 48, clip(-10000*sliceDens ), 96, 255 );
    }
    return shade;
}

__device__ uchar4 volumeRenderShader( float *d_vol, int3 volSize, Ray boxRay, float threshold, int numSteps )
{
    uchar4 shade = make_uchar4( 96, 0, 192, 0 ); // defining background value
    const float dt  = 1.f / numSteps;
    const float len = length( boxRay.d ) / numSteps;

    float accum = 0.0f;
    float3 pos  = boxRay.o;
    float val   = density( d_vol, volSize, pos );

    for ( float t = 0.0f; t < 1.0f; t += dt )
    {
        if( val - threshold < 0.0f )
            accum += fabsf( val - threshold ) * len;

        pos = paramRay( boxRay, t );
        val = density( d_vol, volSize, pos );
    }

    if( clip( accum ) > 0.0f )
        shade.y = clip( accum );

    return shade;
}


__device__ uchar4 rayCastShader( float *d_vol, int3 volSize, Ray boxRay, float dist )
{
    uchar4 shade = make_uchar4( 96, 0, 192, 0 );
    float3 pos   = boxRay.o;

    float  len   = length( boxRay.d );
    float  t     = 0.0f;
    float  f     = density( d_vol, volSize, pos );
    while ( f > dist + EPS && t < 1.0f )
    {
        f   = density( d_vol, volSize, pos );
        t  += ( f - dist ) / len;
        pos = paramRay( boxRay, t );
        f   = density( d_vol, volSize, pos );
    }

    if( t < 1.0f )
    {
        const float3 ux = make_float3( 1, 0, 0 ), uy = make_float3( 0, 1, 0 ), uz = make_float3( 0, 0, 1 );

        float3 grad = { ( density( d_vol, volSize, pos + EPS*ux ) - density( d_vol, volSize, pos ) ) / EPS,
                        ( density( d_vol, volSize, pos + EPS*uy ) - density( d_vol, volSize, pos ) ) / EPS,
                        ( density( d_vol, volSize, pos + EPS*uz ) - density( d_vol, volSize, pos ) ) / EPS
                      };

        float intensity = -dot( normalize( boxRay.d ), normalize( grad ) );
        shade = make_uchar4( 255 * intensity, 0, 0, 255 );
    }

    return shade;
}
